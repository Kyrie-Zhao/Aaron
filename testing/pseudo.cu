#include "hip/hip_runtime.h"
__global__ convKernel(input,output,Kweight,bias):
    output = 0;
    bind(output[.],blockIdx)
    bind(output[,],threadIdx)
    for i in range(0,.,.)
        for j in range(0,.,.)
            .. for z in range(0,.,.)
                    output[] = input[..]*weight[..]
        
    return output

__global__ pooling(buff):
    ...

__global__ dense(input,output,Kweight,bias):
    ...

# Host Codes
__host__ inference():
    memorycpyH2D(input,input_host,input_size) # copy input to device
    convKernel<<<grid,block>>>(input,output,Kweight,bias)
    pooling<<<..>>>(..)
    ... # Launch other kernels in DNN
    dense <<<..>>> (..)
    memcpyD2H(output_host, out, output_size) # copy output to host


